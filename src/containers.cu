#include "containers.cuh"

namespace containers {

CudaStream::CudaStream(unsigned int flags) {
  hipStreamCreateWithFlags(&m_stream, flags);
}

CudaStream::~CudaStream() { hipStreamDestroy(m_stream); }

void CudaStream::sync() { hipStreamSynchronize(m_stream); }

hipStream_t CudaStream::operator()() { return m_stream; }

} // namespace containers
