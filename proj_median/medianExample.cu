#include "hip/hip_runtime.h"
#include "median.cuh"

#include "timer.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <iostream>
#include <limits>
#include <vector>

#include <algorithm>

template <typename T, int maxLength> void testKernel(int numTests) {
  // Initialise inputs to all maximum values
  std::vector<T> input(maxLength * numTests, std::numeric_limits<T>::max());
  std::vector<int> inputLengths(numTests);

  // Randomise a few lengths
  for (int i = 0; i < numTests; i++) {
    int l = 0;
    // Don't want 0-length vectors
    while (l == 0)
      l = rand() % maxLength;

    inputLengths[i] = l;
  }

  // Now randomise values up to the length
  for (int i = 0; i < numTests; ++i) {
    for (int j = 0; j < inputLengths[i]; ++j)
      input[i * maxLength + j] = rand() % 1000;
  }
  printf("Randomised values.\n");

  // Run our kernel
  thrust::device_vector<T> d_input(input);
  thrust::device_vector<int> d_inputLengths(inputLengths);
  thrust::device_vector<T> d_medians(numTests);

  // Simple cub sorting
  {
    const int numThreads = 128;
    const int ELEM_PER_THREAD = 1;
    const int numBlocks = numTests;
    printf("Starting kernel, %d blocks, %d threads.\n", numBlocks, numThreads);
    blockwise_median_kernel<T, numThreads, ELEM_PER_THREAD>
        <<<numBlocks, numThreads>>>(d_input.data().get(), numTests, maxLength,
                                    d_inputLengths.data().get(),
                                    d_medians.data().get());
    printf("Kernel complete\n");
  }

  // Custom quickselect
  {
    thrust::device_vector<int> d_n(numTests);
    thrust::transform(d_inputLengths.begin(), d_inputLengths.end(), d_n.begin(),
                      [] __device__(int x) { return x / 2; });
    const int numThreads = 128;
    const int numBlocks = numTests;
    // const int shmem = sizeof(T) * maxLength * 2 + 2 * sizeof(int);
    // blockwise_quickselect_kernel<T><<<numBlocks, numThreads, shmem>>>(
    blockwise_quickselect_kernel<T, maxLength><<<numBlocks, numThreads>>>(
        d_input.data().get(), numTests, maxLength, d_inputLengths.data().get(),
        d_n.data().get(), d_medians.data().get());
  }

  thrust::host_vector<T> h_medians = d_medians;
  printf("Copied kernel results back\n");

  // Now run the original data with CPU nth_element
  std::vector<T> medianChecks(h_medians.size());
  {
    HighResolutionTimer timer;
    for (int i = 0; i < numTests; i++) {
      std::nth_element(input.begin() + i * maxLength,
                       input.begin() + i * maxLength + inputLengths[i] / 2,
                       input.begin() + i * maxLength + inputLengths[i]);
      medianChecks[i] = input[i * maxLength + inputLengths[i] / 2];
    }
  }

  for (int i = 0; i < numTests; i++) {
    if (h_medians[i] != medianChecks[i]) {
      std::cout << "Median check failed at index " << i << "!" << std::endl;
      std::cout << "Input length: " << inputLengths[i] << std::endl;
      std::cout << "Median, index " << i << ": " << h_medians[i] << " vs "
                << medianChecks[i] << std::endl;
    }
  }
}

int main() {
  printf("Median kernel tests\n");

  testKernel<unsigned short, 100>(10000);

  return 0;
}
