#include "hip/hip_runtime.h"
#include "selectors.cuh"

#include <cstdlib>
#include <limits>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

int main() {
  int rows = 64;
  int cols = 64;
  thrust::host_vector<unsigned short> h_input(rows * cols);
  thrust::host_vector<unsigned short> h_conditional(rows * cols);
  for (int i = 0; i < rows; ++i) {
    for (int j = 0; j < cols; ++j) {
      h_conditional[i * cols + j] = std::rand() % 10;
      h_input[i * cols + j] = std::rand() % 10;
    }
  }
  thrust::device_vector<unsigned short> d_input = h_input;
  thrust::device_vector<unsigned short> d_conditional = h_conditional;

  // Allocate sufficient output for ROI
  const unsigned int roiStartRow = 5, roiStartCol = 5;
  const unsigned int roiNumRows = 32, roiNumCols = 32;
  const unsigned int roiRowStride = 1, roiColStride = 2;
  // Count the number of rows/cols actually used
  const unsigned int roiNumStridedRows =
      roiNumRows / roiRowStride + (roiNumRows % roiRowStride == 0 ? 0 : 1);
  const unsigned int roiNumStridedCols =
      roiNumCols / roiColStride + (roiNumCols % roiColStride == 0 ? 0 : 1);
  thrust::device_vector<unsigned short> d_out(roiNumStridedRows *
                                              roiNumStridedCols);
  printf("roiNumStridedRows = %u, roiNumStridedCols = %u, d_out size = %zd\n",
         roiNumStridedRows, roiNumStridedCols, d_out.size());

  dim3 NUM_THREADS(32, 8);
  dim3 NUM_BLKS(roiNumCols / NUM_THREADS.x + 1, roiNumRows / NUM_THREADS.y + 1);
  maximum_conditioned_downsampling_kernel<unsigned short>
      <<<NUM_BLKS, NUM_THREADS>>>(
          d_input.data().get(), d_conditional.data().get(), 7, rows, cols,
          roiStartRow, roiStartCol, roiNumRows, roiNumCols, roiRowStride,
          roiColStride, d_out.data().get(), roiNumStridedRows,
          roiNumStridedCols);

  thrust::host_vector<unsigned short> h_out = d_out;

  printf("==================\nOutput:\n");
  for (unsigned int i = 0; i < roiNumStridedRows; ++i) {
    for (unsigned int j = 0; j < roiNumStridedCols; ++j) {
      auto out = h_out[i * roiNumStridedCols + j];
      if (out == std::numeric_limits<unsigned short>::max())
        printf("* ");
      else
        printf("%hu ", out);
    }
    std::cout << std::endl;
  }

  printf("==================\nInput:\n");
  for (unsigned int i = 0; i < roiNumStridedRows; ++i) {
    for (unsigned int j = 0; j < roiNumStridedCols; ++j) {
      printf("%hu ", h_input[(i * roiRowStride + roiStartRow) * cols +
                             j * roiColStride + roiStartCol]);
    }
    std::cout << std::endl;
  }

  printf("==================\nConditional:\n");
  for (unsigned int i = 0; i < roiNumStridedRows; ++i) {
    for (unsigned int j = 0; j < roiNumStridedCols; ++j) {
      printf("%hu ", h_conditional[(i * roiRowStride + roiStartRow) * cols +
                                   j * roiColStride + roiStartCol]);
    }
    std::cout << std::endl;
  }

  return 0;
}
